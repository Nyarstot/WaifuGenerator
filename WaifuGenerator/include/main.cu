#include <iostream>
#include <hip/hip_runtime.h>




__global__ void AddIntsCUDA(int* a, int* b) 
{
    a[0] += b[0];
}

int main()
{
    int a = 5, b = 9;
    int* d_a, * d_b;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    AddIntsCUDA <<<1, 1>>> (d_a, d_b);

    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Answer: " << a;

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}