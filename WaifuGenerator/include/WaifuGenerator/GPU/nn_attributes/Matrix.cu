#include "Matrix.hh"

#include "../cuda/cu_core.hh"

WaifuGPU::Matrix::Matrix()
{
    m_Rows = 0;
    m_Cols = 0;

    m_Buffer = nullptr;
}

WaifuGPU::Matrix::Matrix(const size_t& size) : m_Rows(size), m_Cols(size)
{
    this->init(m_Rows, m_Cols);
}

WaifuGPU::Matrix::~Matrix()
{
    hipFree(m_Buffer);
    m_Buffer = nullptr;
}

void WaifuGPU::Matrix::init(const size_t& rows, const size_t& cols)
{
    m_Rows = rows;
    m_Cols = cols;

    if ((m_Rows > 0) && (m_Cols > 0)) {
        hipMalloc((void**)&m_Buffer, m_Rows * m_Cols * sizeof(double));
        hipMemset(m_Buffer, 0, m_Rows * m_Cols * sizeof(double));
    }
}

void WaifuGPU::Matrix::rand()
{
    double rand_value = 0;

    for (size_t i{}; i < m_Rows; i++) {
        for (size_t j{}; j < m_Cols; j++) {
            rand_value = ((std::rand() % 100)) * 0.03 / (m_Rows + 35);
            hipMemcpy(&m_Buffer[j * m_Rows + i], &rand_value, sizeof(double), hipMemcpyHostToDevice);
        }
    }
}

size_t WaifuGPU::Matrix::get_rows()
{
    return this->m_Rows;
}

size_t WaifuGPU::Matrix::get_cols()
{
    return this->m_Cols;
}

void WaifuGPU::Matrix::debug_print()
{
    double tmp;

    for (size_t i{}; i < m_Rows; i++) {
        for (size_t j{}; j < m_Cols; j++) {
            hipMemcpy(&tmp, &m_Buffer[j * m_Rows + i], sizeof(double), hipMemcpyDeviceToHost);
            std::cout << tmp << " ";
        }
        std::cout << "\n";
    }
}